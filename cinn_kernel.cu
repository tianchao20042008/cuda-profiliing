#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#define CINN_WITH_CUDA
#define CINN_CUDA_FP16
#include <stdio.h>
#include "float16.h"
using cinn::common::float16;
using cinn::common::half4;
using cinn::common::half8;
using cinn::common::float8;
#include <iostream>

constexpr int N=256, H=112, W=112, C=64;
constexpr int REDUCE_NUMEL = N*H*W;

#include "no_zero_opt.cuh"
#include "zero_opt.cuh"

int main() {
  hipSetDevice(0);
  // 256x56x56x256
	const float16* elementwise_add_1____GRAD = nullptr;
  size_t size_elementwise_add_1____GRAD = sizeof(float16) * 256 * 56 * 56 * 256;
  hipMalloc((void**)&(elementwise_add_1____GRAD), size_elementwise_add_1____GRAD);
  // 256x56x56x256
  const float16* relu_3__tmp_0____GRAD____RENAME____block0____1 = nullptr;
  size_t size_relu_3__tmp_0____GRAD____RENAME____block0____1 = sizeof(float16) * 256 * 56 * 56 * 256;
  hipMalloc((void**)&(relu_3__tmp_0____GRAD____RENAME____block0____1), size_relu_3__tmp_0____GRAD____RENAME____block0____1);
  // 256x56x56x256
  const float16* relu_3__tmp_0 = nullptr;
  size_t size_relu_3__tmp_0 = sizeof(float16) * 256 * 56 * 56 * 256;
  hipMalloc((void**)&(relu_3__tmp_0), size_relu_3__tmp_0);
  // 256x56x56x256
  const float16* conv2d_57__tmp_0 = nullptr;
  size_t size_conv2d_57__tmp_0 = sizeof(float16) * 256 * 56 * 56 * 256;
  hipMalloc((void**)&(conv2d_57__tmp_0), size_conv2d_57__tmp_0);
  // 256x56x56x256
  const float16* conv2d_56__tmp_0 = nullptr;
  size_t size_conv2d_56__tmp_0 = sizeof(float16) * 256 * 56 * 56 * 256;
  hipMalloc((void**)&(conv2d_56__tmp_0), size_conv2d_56__tmp_0);
  // 256
  const float* batch_norm_3__tmp_0 = nullptr;
  size_t size_batch_norm_3__tmp_0 = sizeof(float) * 256;
  hipMalloc((void**)&(batch_norm_3__tmp_0), size_batch_norm_3__tmp_0);
  // 256
  const float* batch_norm_4__tmp_0 = nullptr;
  size_t size_batch_norm_4__tmp_0 = sizeof(float) * 256;
  hipMalloc((void**)&(batch_norm_4__tmp_0), size_batch_norm_4__tmp_0);
  // 896x896x256
  float* var_15740 = nullptr;
  size_t size_var_15740 = sizeof(float) * 896 * 896 * 256;
  hipMalloc((void**)&(var_15740), size_var_15740);
  // 896x256
  float* var_15941 = nullptr;
  size_t size_var_15941 = sizeof(float) * 896 * 256;
  hipMalloc((void**)&(var_15941), size_var_15941);
  // 896x256
  float* var_15938 = nullptr;
  size_t size_var_15938 = sizeof(float) * 896 * 256;
  hipMalloc((void**)&(var_15938), size_var_15938);
  // 896x256
  float* var_15932 = nullptr;
  size_t size_var_15932 = sizeof(float) * 896 * 256;
  hipMalloc((void**)&(var_15932), size_var_15932);
  // 256x56x56x256
  float* var_15742 = nullptr;
  size_t size_var_15742 = sizeof(float) * 256 * 56 * 56 * 256;
  hipMalloc((void**)&(var_15742), size_var_15742);
  // 896x256
  float* var_15935 = nullptr;
  size_t size_var_15935 = sizeof(float) * 896 * 256;
  hipMalloc((void**)&(var_15935), size_var_15935);
  for (int i = 0; i < 20; ++i) {
    no_zero_opt<<<224, 1024>>>(
      // 256x56x56x256
      elementwise_add_1____GRAD,
      // 256x56x56x256
      relu_3__tmp_0____GRAD____RENAME____block0____1,
      // 256x56x56x256
      relu_3__tmp_0,
      // 256x56x56x256
      // conv2d_57__tmp_0,
      // 256x56x56x256
      conv2d_56__tmp_0,
      // 256
      batch_norm_3__tmp_0,
      // 256
      // batch_norm_4__tmp_0,
      // 896x896x256
      // float* __restrict__ var_15740,
      // 896x256
      var_15941,
      // 896x256
      var_15938,
      // 896x256
      var_15932,
      // 256x56x56x256
      var_15742//,
      // 896x256
      //float* __restrict__ var_15935
    );
    zero_opt<<<224, 1024>>>(
      // 256x56x56x256
      elementwise_add_1____GRAD,
      // 256x56x56x256
      relu_3__tmp_0____GRAD____RENAME____block0____1,
      // 256x56x56x256
      relu_3__tmp_0,
      // 256x56x56x256
      // conv2d_57__tmp_0,
      // 256x56x56x256
      conv2d_56__tmp_0,
      // 256
      batch_norm_3__tmp_0,
      // 256
      // batch_norm_4__tmp_0,
      // 896x896x256
      // float* __restrict__ var_15740,
      // 896x256
      var_15941,
      // 896x256
      var_15938,
      // 896x256
      var_15932,
      // 256x56x56x256
      var_15742//,
      // 896x256
      //float* __restrict__ var_15935
    );
  }
  hipDeviceSynchronize();
  printf("%s\n", hipGetErrorString(hipGetLastError()));
  return 0;
}
